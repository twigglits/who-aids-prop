#include <hip/hip_runtime.h>
#include <iostream>

#define N 512  // Assuming matrix size is 512x512

__global__ void multiply_matrices_gpu(const float* A, const float* B, float* C)  
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0.0f;

    if (row < N && col < N) {
        for (int k = 0; k < N; ++k) {
            sum += A[row * N + k] * B[k * N + col];
        }
        C[row * N + col] = sum;
    }
}

int main() {
    // Host matrices
    float *h_A, *h_B, *h_C;
    // Device matrices
    float *d_A, *d_B, *d_C;

    size_t bytes = N * N * sizeof(float);

    // Allocate memory on host
    h_A = (float*)malloc(bytes);
    h_B = (float*)malloc(bytes);
    h_C = (float*)malloc(bytes);

    // Initialize matrices A and B
    for (int i = 0; i < N * N; i++) {
        h_A[i] = static_cast<float>(i);
        h_B[i] = static_cast<float>(i);
    }

    // Allocate memory on device
    hipMalloc(&d_A, bytes);
    hipMalloc(&d_B, bytes);
    hipMalloc(&d_C, bytes);

    // Copy data from host to device
    hipMemcpy(d_A, h_A, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, bytes, hipMemcpyHostToDevice);

    // Define block size and grid size
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((N + threadsPerBlock.x - 1) / threadsPerBlock.x, 
                       (N + threadsPerBlock.y - 1) / threadsPerBlock.y);

    // Launch the kernel
    multiply_matrices_gpu<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C);

    // Wait for GPU to finish
    hipDeviceSynchronize();

    // Copy result back to host
    hipMemcpy(h_C, d_C, bytes, hipMemcpyDeviceToHost);

    // Print a portion of the result matrix C
    std::cout << "Result matrix C[0][0]: " << h_C[0] << std::endl;

    // Free memory
    free(h_A);
    free(h_B);
    free(h_C);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
