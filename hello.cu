
#include <hip/hip_runtime.h>
#include <iostream>

// Kernel function to print "Hello, World!" from the GPU
__global__ void helloWorldKernel() {
    printf("Hello, World from GPU!\n");
}

int main() {
    // Launch the kernel with a single thread
    helloWorldKernel<<<1, 1>>>();

    // Wait for the GPU to finish before accessing on host
    hipDeviceSynchronize();

    std::cout << "Hello, World from CPU!" << std::endl;

    return 0;
}
